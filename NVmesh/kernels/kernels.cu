#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "utils.h"
#include "error.h"
#include "common.h"

#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


// template<int N>
// __global__ void copyKernel(unsigned long long loopCount, uint4 *dst, uint4 *src, size_t chunkSizeInElement, unsigned int totalThreadCount)
// {
//     unsigned long long from = blockDim.x * blockIdx.x + threadIdx.x;
//     dst += from;
//     src += from;
//     unsigned long long chunkSizeModifyInElement = chunkSizeInElement / N;
//     unsigned long long chunkSizeleftInElement = chunkSizeInElement - chunkSizeModifyInElement * N;
//     for (unsigned long long i = 0; i < loopCount; i++) {
//         uint4 * cdst = dst;
//         uint4 * csrc = src;
//         for (unsigned int j = 0; j < chunkSizeModifyInElement; j++) {
//             int4 pipe[N];
//             #pragma unroll
//             for (unsigned int k = 0; k < N; k++) {
//                 pipe[k] = ld_nc_global(csrc);
//                 csrc += totalThreadCount;
//             }
//             #pragma unroll
//             for (unsigned int k = 0; k < N; k++) {
//                 st_nc_global(cdst, pipe[k]);
//                 cdst += totalThreadCount;
//             }
//         }
//         for (unsigned int j = 0; j < chunkSizeleftInElement; j++) {
//             st_nc_global(cdst, ld_nc_global(csrc));
//             cdst += totalThreadCount;
//             csrc += totalThreadCount;
//         }
//     }
// }

__global__ void copyKernel_fallback(unsigned long long loopCount, uint4 *dst, uint4 *src, size_t chunkSizeInElement, unsigned int totalThreadCount) {
    unsigned long long from = blockDim.x * blockIdx.x + threadIdx.x;
    dst += from;
    src += from;
    unsigned long long chunkSizeModifyInElement = chunkSizeInElement / totalThreadCount;
    for (unsigned int i = 0; i < loopCount; i++) {
        uint4* cdst = dst;
        uint4* csrc = src;
        for (unsigned int j = 0; j < chunkSizeModifyInElement; j++) {
            st_nc_global(cdst, ld_nc_global(csrc));
            cdst += totalThreadCount;
            csrc += totalThreadCount;
        }
    }
}
#define FOR_EACH_PIPES(M)  M(1) M(2) M(4) M(8) M(12) M(16) M(32)

// #    kernel type 1
// #    chunkSizeIntElement = Number of uint4 per totalThreadCount
// #    dst
// #    |---------------------------------------------------------| BufferSize
// #    chunk1        chunk2
// #    |-------------|-------------|......
// #      |             |
// #      cdst          cdst += totalThreadCount   

#define DEF_COPY(N)                                                                 \
__global__ void copyKernel_##N##pipes(unsigned long long loopCount, uint4 *dst, uint4 *src, size_t chunkSizeInElement, unsigned int totalThreadCount) { \
    unsigned long long from = blockDim.x * blockIdx.x + threadIdx.x;                \
    dst += from;                                                                    \
    src += from;                                                                    \
    unsigned long long chunkSizeModifyInElement =  chunkSizeInElement / N;          \
    unsigned long long chunkSizeleftInElement =  chunkSizeInElement - chunkSizeModifyInElement * N; \
    for (unsigned int i = 0; i < loopCount; i++) {                                  \
        uint4* cdst = dst;                                                          \
        uint4* csrc = src;                                                          \
        for (unsigned int j = 0; j < chunkSizeModifyInElement; j++) {               \
            int4 pipe[N];                                                           \
            #pragma unroll                                                          \
            for (int k = 0; k < N; k++) {                                           \
                pipe[k] = ld_nc_global(csrc);                                       \
                csrc += totalThreadCount;                                           \
            }                                                                       \
            #pragma unroll                                                          \
            for (int k = 0; k < N; k++) {                                           \
                st_nc_global(cdst, pipe[k]);                                        \
                cdst += totalThreadCount;                                           \
            }                                                                       \
        }                                                                           \
        for (unsigned int j = 0; j < chunkSizeleftInElement; j++) {                 \
            st_nc_global(cdst, ld_nc_global(csrc));                                 \
            cdst += totalThreadCount;                                               \
            csrc += totalThreadCount;                                               \
        }                                                                           \
    }                                                                               \
}
FOR_EACH_PIPES(DEF_COPY)





__global__ void spinKernelDevice(volatile int *latch, const unsigned long long timeoutClocks) {
    register unsigned long long endTime = _read_globaltimer() + timeoutClocks;
    while (!*latch) {
        if (timeoutClocks != ~0ULL && _read_globaltimer() > endTime) {
            break;
        }
    }
}

hipError_t spinKernel(volatile int *latch, hipStream_t stream, unsigned long long timeoutMs) {
    int clocksPerMs = 0;
    hipCtx_t ctx;
    hipDevice_t dev;

    CU_ASSERT(cuStreamGetCtx(stream, &ctx));
    CU_ASSERT(hipCtxGetDevice(&dev));

    CU_ASSERT(hipDeviceGetAttribute(&clocksPerMs, hipDeviceAttributeClockRate, dev));

    unsigned long long timeoutClocks = clocksPerMs * timeoutMs;

    spinKernelDevice<<<1, 1, 0, stream>>>(latch, timeoutClocks);

    return hipSuccess;
}

__global__ void spinKernelDeviceMultistage(volatile int *latch1, volatile int *latch2, const unsigned long long timeoutClocks) {
    if (latch1) {
        register unsigned long long endTime = clock64() + timeoutClocks;
        while (!*latch1) {
            if (timeoutClocks != ~0ULL && clock64() > endTime) {
                break;
            }
        }

        *latch2 = 1;
    }
    
    register unsigned long long endTime = clock64() + timeoutClocks;
    while (!*latch2) {
        if (timeoutClocks != ~0ULL && clock64() > endTime) {
            break;
        }
    }
}

hipError_t spinKernelMultistage(volatile int *latch1, volatile int *latch2, hipStream_t stream, unsigned long long timeoutMs) {
    ASSERT(latch2 != nullptr);

    unsigned long long timeoutClocks = rank::getLocalClockRate() * timeoutMs;
    spinKernelDeviceMultistage<<<1, 1, 0, stream>>>(latch1, latch2, timeoutClocks);
    CUDA_ASSERT(hipPeekAtLastError());

    return hipSuccess;
}

__global__ void patternFillKernel(uint4* dst, int seed, size_t bufferSize, int groupId, int groupSize) {
    unsigned long long from = blockDim.x * blockIdx.x + threadIdx.x;
    size_t totalThreadCount = gridDim.x * blockDim.x;
    char* dstEnd = ((char *) dst) + bufferSize;
    dst += from;

    hiprandStateXORWOW_t state;
    hiprand_init(seed, 0, from, &state);

    for (int i = 0; i < groupSize; i++) {
        hiprand(&state);
    }

    while ((char *) dst < dstEnd) {
        *dst = hiprand(&state);
        dst += totalThreadCount;

        for (int i = 0; i < groupSize - 1; i++) {
            hiprand(&state);
        }
    }
}

void memsetBuffer(void *ptr, int seed, size_t size, hipStream_t stream, int groupId, int groupSize) {
    dim3 gridDim(rank::getLocalMultiprocessorCount(), 1, 1);
    dim3 blockDim(numThreadPerBlock, 1, 1);
    patternFillKernel<<<gridDim, blockDim, 0, stream>>>((uint *)ptr, seed, size, groupId, groupSize);
    CUDA_ASSERT(hipPeekAtLastError());
}

void zeroOutBuffer(void *ptr, size_t size, hipStream_t stream) {
    CU_ASSERT(hipMemsetD8Async((hipDeviceptr_t) ptr, 0, size, stream));
}

void memsetBuffer(void *ptr, int seed, size_t size, hipStream_t stream, CopyType copyType, MemoryPurpose memoryPurpose) {
    // if (copyType == COPY_TYPE_MULTICAST_LD_REDUCE) {
    //     memsetBuffer(ptr, seed, size, stream, MPIWrapper::getWorldRank(), MPIWrapper::getWorldSize());
    // } else if (copyType == COPY_TYPE_MULTICAST_RED_ALL) {
    //     if (memoryPurpose == MemoryPurpose::MEMORY_SOURCE) {
    //         memsetBuffer(ptr, seed, size, stream, MPIWrapper::getWorldRank(), MPIWrapper::getWorldSize());
    //     } else {
    //         zeroOutBuffer(ptr, size, stream);
    //     }
    // } else if (copyType == COPY_TYPE_MULTICAST_RED_SINGLE) {
    //     if (memoryPurpose == MemoryPurpose::MEMORY_SOURCE) {
    //         memsetBuffer(ptr, seed, size, stream, 0, 1);
    //     } else {
    //         zeroOutBuffer(ptr, size, stream);
    //     }
    // } else {
    //     memsetBuffer(ptr, seed, size, stream, 0, 1);
    // }
    memsetBuffer(ptr, seed, size, stream, 0, 1);
}

// 验证错误
__global__ void patternCheckKernel(uint* buffer, int seed, size_t bufferSize, unsigned long long *errorCount, int groupSize, int multiplier) {
    uint* originalBuffer = buffer;
    unsigned long long threadId = blockDim.x * blockIdx.x + threadIdx.x;
    size_t totalThreadCount = gridDim.x * blockDim.x;
    char* bufferEnd = ((char *) buffer) + bufferSize;
    buffer += threadId;

    hiprandStateXORWOW_t state;
    hiprand_init(seed, 0, threadId, &state);

    while ((char *) buffer < bufferEnd) {
        uint expectedValue = 0;

        for (int i = 0; i < groupSize; i++) {
            // overflow for uint is well defined
            expectedValue += hiprand(&state);
        }

        expectedValue *= multiplier;

        uint actualValue = *buffer;
        if (actualValue != expectedValue) {
            printf("Error found at byte offset %llu: expected %u but got %u\n", (char *) buffer - (char *) originalBuffer, expectedValue, actualValue);
            atomicAdd(errorCount, 1);
            // Only report one error per thread to avoid spamming prints
            break;
        }
        buffer += totalThreadCount;
    }
}

unsigned long long checkBuffer(void *ptr, int seed, size_t size, hipStream_t stream, int groupSize, int multiplier = 1) {
    unsigned long long *errorCount;
    CU_ASSERT(hipMalloc((hipDeviceptr_t *) &errorCount, sizeof(*errorCount)));
    CU_ASSERT(hipMemsetD8((hipDeviceptr_t) errorCount, 0, sizeof(*errorCount)));

    dim3 gridDim(NvLoom::getLocalMultiprocessorCount(), 1, 1);
    dim3 blockDim(numThreadPerBlock, 1, 1);
    patternCheckKernel<<<gridDim, blockDim, 0, stream>>>((uint *)ptr, seed, size, errorCount, groupSize, multiplier);
    CUDA_ASSERT(hipPeekAtLastError());
    CU_ASSERT(hipStreamSynchronize(stream));

    unsigned long long errorCountCopy;
    CU_ASSERT(cuMemcpy((hipDeviceptr_t) &errorCountCopy, (hipDeviceptr_t) errorCount, sizeof(errorCountCopy)));

    CU_ASSERT(hipFree((hipDeviceptr_t) errorCount));

    return errorCountCopy;
}

unsigned long long checkBuffer(void *ptr, int seed, size_t size, hipStream_t stream, CopyType copyType, int iterations) {
    // if (copyType == COPY_TYPE_MULTICAST_LD_REDUCE) {
    //     return checkBuffer(ptr, seed, size, stream, MPIWrapper::getWorldSize(), 1);
    // } else if (copyType == COPY_TYPE_MULTICAST_RED_ALL) {
    //     return checkBuffer(ptr, seed, size, stream, MPIWrapper::getWorldSize(), iterations);
    // } else if (copyType == COPY_TYPE_MULTICAST_RED_SINGLE) {
    //     return checkBuffer(ptr, seed, size, stream, 1, iterations);
    // } else {
    //     return checkBuffer(ptr, seed, size, stream, 1);
    // }
    return checkBuffer(ptr, seed, size, stream, 1);
}

void preloadKernels(int localDevice) {
    hipFuncAttributes unused;
    hipSetDevice(localDevice);
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&copyKernel_1pipes));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&copyKernel_2pipes));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&copyKernel_4pipes));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&copyKernel_8pipes));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&copyKernel_12pipes));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&copyKernel_16pipes));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&copyKernel_32pipes));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&copyKernel_fallback));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&spinKernelDevice));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&spinKernelDeviceMultistage));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&patternFillKernel));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&patternCheckKernel));
}